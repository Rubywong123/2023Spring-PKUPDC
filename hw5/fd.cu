#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include "CycleTimer.h"

// TO run with tolerance 1e-4 and 4x4 loop currents
//    ./fd 4 1e-4

#define PI 3.14159265359
#define MAX(a,b) (((a)>(b))?(a):(b))

//kernel function
__global__ void compute_unew(double* u, double* f, double* unew, int N, double w, double invD){
  int xid = blockIdx.x * blockDim.x + threadIdx.x;
  int yid = blockIdx.y * blockDim.y + threadIdx.y;
  if(xid < 1 || xid >N || yid < 1 || yid > N)return;
  int id = xid + (yid) * (N+2);
  
  const double Ru = -u[id-(N+2)]-u[id+(N+2)]-u[id-1]-u[id+1];
	const double rhs = invD*(f[id]-Ru);
	const double oldu = u[id];
  // similar to gradient descent?
	const double newu = w*rhs + (1.0-w)*oldu;
  unew[id] = newu;
  __syncthreads();
}

// solve for solution vector u
// host function
int solve(const int N, const double tol, double * u, double * f){

  double start = CycleTimer::currentSeconds();
  double *unew = (double*)calloc((N+2)*(N+2),sizeof(double));
  size_t size = (N+2)*(N+2) * sizeof(double);

  //device memory
  double* d_u = NULL;
  hipMalloc((void**)&d_u, size);
  double* d_f = NULL;
  hipMalloc((void**)&d_f, size);
  double* d_unew = NULL;
  hipMalloc((void**)&d_unew, size);

  double malloc_time = CycleTimer::currentSeconds() - start;
  printf("Malloc time: %.4f\n", malloc_time);

  hipMemcpy(d_f, f, size, hipMemcpyHostToDevice);
  double res2 = 1.0;
  unsigned int iter = 0;
  double w = 1.0;
  double invD = 1./4.;  // factor of h cancels out
  while(res2>tol*tol){

    res2 = 0.0;

    // copy from host to device
    // start = CycleTimer::currentSeconds();
    hipMemcpy(d_u, u, size, hipMemcpyHostToDevice);
    // double copy_time = CycleTimer::currentSeconds() - start;
    // printf("Copy_time: %.4f\n", copy_time);

    //setup device blocks & threads, then call kernel function
    // start = CycleTimer::currentSeconds();
    dim3 threadPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadPerBlock.x - 1) / threadPerBlock.x,( N + threadPerBlock.y - 1) / threadPerBlock.y);
    compute_unew <<< blocksPerGrid, threadPerBlock >>> (d_u, d_f, d_unew, N, w, invD);
    
    hipGetLastError();
    // double compute_time = CycleTimer::currentSeconds() - start;
    // printf("Computation time: %.4f\n", compute_time);

    //copy from device to host
    hipMemcpy(unew, d_unew, size, hipMemcpyDeviceToHost);

    // start = CycleTimer::currentSeconds();
    for(int i=1; i<=N; ++i){
      for(int j=1; j<=N; ++j){
        int id = i + j * (N+2);
        double newu = unew[id];
        double oldu = u[id];
	      res2 += (newu-oldu)*(newu-oldu);
      }
    }
    // double res_time = CycleTimer::currentSeconds() - start;
    // printf("Compute res time: %.4f\n", res_time);

    // start = CycleTimer::currentSeconds();
    for (int i = 0; i < (N+2)*(N+2); ++i){
      u[i] = unew[i];
    }
    // double update_time =  CycleTimer::currentSeconds() - start;
    // printf("Update time: %.4f\n", update_time);

    ++iter;
    if(!(iter%500)){
      printf("at iter %d: residual = %g\n", iter, sqrt(res2));
    }
  }

  hipFree(d_f);
  hipFree(d_u);
  hipFree(d_unew);

  // free(unew);

  return iter;
}

int main(int argc, char **argv){
  
  if(argc!=3){
    printf("Usage: ./main N tol\n");
    exit(-1);
  }
  
  int N = atoi(argv[1]);
  double tol = atof(argv[2]);

  // flatten array
  double *u = (double*) calloc((N+2)*(N+2), sizeof(double));
  double *f = (double*) calloc((N+2)*(N+2), sizeof(double));
  double h = 2.0/(N+1);
  for (int i = 0; i < N+2; ++i){
    for (int j = 0; j < N+2; ++j){
      const double x = -1.0 + i*h;
      const double y = -1.0 + j*h;
      f[i + j*(N+2)] = sin(PI*x)*sin(PI*y) * h*h;
    }
  }
  double err = 0.0;
  for (int i = 0; i < (N+2)*(N+2); ++i){
    // solution: 1/2PI^2 * sin(PI*x) * sin(PI*y)
    // if (fabs(u[i] - f[i]/(h*h*2.0*PI*PI)) > err){
    //   printf("%d, %.4f, %.4f\n", i, fabs(u[i] - f[i]/(h*h*2.0*PI*PI)), err);
    // }
    err = MAX(err,fabs(u[i] - f[i]/(h*h*2.0*PI*PI)));
  }
  printf("Max error: %lg\n", err);

  //u: random array at initial state
  double start = CycleTimer::currentSeconds();
  int iter = solve(N, tol, u, f);
  double solve_time = CycleTimer::currentSeconds() - start;
  printf("Solve time: %.4f\n", solve_time);

  err = 0.0;
  for (int i = 0; i < (N+2)*(N+2); ++i){
    // if (fabs(u[i] - f[i]/(h*h*2.0*PI*PI)) > err){
    //   printf("%d, %.4f, %.4f\n", i, fabs(u[i] - f[i]/(h*h*2.0*PI*PI)), err);
    // }
    // solution: 1/2PI^2 * sin(PI*x) * sin(PI*y)
    err = MAX(err,fabs(u[i] - f[i]/(h*h*2.0*PI*PI)));
  }
  
  printf("Iters: %d\n", iter);
  printf("Max error: %lg\n", err);
  printf("Memory usage: %lg GB\n", (N+2)*(N+2)*sizeof(double)/1.e9);
  
  free(u);
  free(f);  

}
  
